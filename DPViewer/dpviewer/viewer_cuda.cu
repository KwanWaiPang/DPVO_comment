#include "hip/hip_runtime.h"
#include "viewer_cuda.h"

#define THREADS 64

#define NUM_BLOCKS(batch_size) ((batch_size + THREADS - 1) / THREADS)

#define GPU_1D_KERNEL_LOOP(k, n) \
  for (size_t k = threadIdx.x; k<n; k += blockDim.x)


__device__ void
actSO3(const float* __restrict__ q, 
       const float* __restrict__ X, 
       float* __restrict__ Y) {
  
  float uv[3];
  uv[0] = 2.0 * (q[1]*X[2] - q[2]*X[1]);
  uv[1] = 2.0 * (q[2]*X[0] - q[0]*X[2]);
  uv[2] = 2.0 * (q[0]*X[1] - q[1]*X[0]);

  Y[0] = X[0] + q[3]*uv[0] + (q[1]*uv[2] - q[2]*uv[1]);
  Y[1] = X[1] + q[3]*uv[1] + (q[2]*uv[0] - q[0]*uv[2]);
  Y[2] = X[2] + q[3]*uv[2] + (q[0]*uv[1] - q[1]*uv[0]);
}

__device__  void
actSE3(const float* __restrict__ t, 
       const float* __restrict__ q, 
       const float* __restrict__ X, 
       float* __restrict__ Y) {
  
  actSO3(q, X, Y);
  Y[3] = X[3];
  Y[0] += X[3] * t[0];
  Y[1] += X[3] * t[1];
  Y[2] += X[3] * t[2];
}

__device__ void
invSE3(const float* __restrict__ t, 
       const float* __restrict__ q, 
       float* __restrict__ tinv, 
       float* __restrict__ qinv) {
  qinv[0] = -q[0];
  qinv[1] = -q[1];
  qinv[2] = -q[2];
  qinv[3] =  q[3];
  
  actSO3(qinv, t, tinv);
  tinv[0] = -tinv[0];
  tinv[1] = -tinv[1];
  tinv[2] = -tinv[2];
}


__global__ void iproj_kernel(const int index, const int nFrames, const float thresh,
    const torch::PackedTensorAccessor32<unsigned char,4,torch::RestrictPtrTraits> images,
    const torch::PackedTensorAccessor32<float,2,torch::RestrictPtrTraits> poses,
    const torch::PackedTensorAccessor32<float,3,torch::RestrictPtrTraits> disps,
    const torch::PackedTensorAccessor32<float,2,torch::RestrictPtrTraits> intrinsics,
    torch::PackedTensorAccessor32<float,2,torch::RestrictPtrTraits> points,
    torch::PackedTensorAccessor32<unsigned char,2,torch::RestrictPtrTraits> colors,
    torch::PackedTensorAccessor32<float,1,torch::RestrictPtrTraits> count)
{

  __shared__ float t[3], t1[3], t2[3];
  __shared__ float q[4], q1[4], q2[4];
  __shared__ float intrinsic[4], intrinsic1[4];

  if (threadIdx.x < 3) {
    t[threadIdx.x] = poses[index][threadIdx.x + 0];
  }

  if (threadIdx.x < 4) {
    q[threadIdx.x] = poses[index][threadIdx.x + 3];
  }

  if (threadIdx.x < 4) {
    intrinsic[threadIdx.x] = 8 * intrinsics[index][threadIdx.x];
  }

  __syncthreads();


  if (threadIdx.x == 0) {
    invSE3(t, q, t1, q1);
  }

  __syncthreads();

  const int ht = disps.size(1);
  const int wd = disps.size(2);

  const int k = blockIdx.x * THREADS + threadIdx.x;

  if (k < ht * wd) {
    float X0[4], X1[4], X2[4];
    const int i = k / wd;
    const int j = k % wd;

    count[k] = 0;

    if ((i < ht - 1) && (j < wd - 1)) {
      const float d = disps[index][i][j];
      const float dx = disps[index][i][j+1] - disps[index][i][j];
      const float dy = disps[index][i+1][j] - disps[index][i][j];

      if (sqrt(dx*dx + dy*dy) > 0.01) {
        count[k] = -100;
      }

      X0[0] = ((float) j - intrinsic[2]) / intrinsic[0];
      X0[1] = ((float) i - intrinsic[3]) / intrinsic[1];
      X0[2] = 1;
      X0[3] = d;

      actSE3(t1, q1, X0, X1);

      points[k][0] = X0[0] / X0[3];
      points[k][1] = X0[1] / X0[3];
      points[k][2] = X0[2] / X0[3];

      colors[k][0] = images[index][2][i][j];
      colors[k][1] = images[index][1][i][j];
      colors[k][2] = images[index][0][i][j];


      for (int jx=0; jx < nFrames; jx++) {
        if (jx == index) continue;

        if (threadIdx.x < 3) {
          t2[threadIdx.x] = poses[jx][threadIdx.x + 0];
        }

        if (threadIdx.x < 4) {
          q2[threadIdx.x] = poses[jx][threadIdx.x + 3];
        }

        if (threadIdx.x < 4) {
          intrinsic1[threadIdx.x] = 8 * intrinsics[jx][threadIdx.x];
        }

        __syncthreads();

        actSE3(t2, q2, X1, X2);

        const float x1 = intrinsic1[0] * (X2[0] / X2[2]) + intrinsic1[2];
        const float y1 = intrinsic1[1] * (X2[1] / X2[2]) + intrinsic1[3];

        const int i1 = static_cast<int>(round(y1));
        const int j1 = static_cast<int>(round(x1));

        if ((i1 >= 0) && (i1 < ht) && (j1 >= 0) && (j1 < wd) && (d > 0.1)) {
          const float z1 = disps[jx][i1][j1];
          const float z2 = X2[3] / X2[2];
          
          if (100 * (max(z1/z2, z2/z1) - 1) < thresh) {
            count[k] += 1;
          }
        }
      }
    }
  }
}


PointCloud backproject_and_filter(
    const int index,
    const int nFrames,
    const float thresh,
    const bool showForeground,
    const bool showBackground,
    const torch::Tensor images,
    const torch::Tensor poses,
    const torch::Tensor disps,
    const torch::Tensor masks,
    const torch::Tensor intrinsics) 
{
  const int ht = disps.size(1);
  const int wd = disps.size(2);

  const int nPoints = ht * wd;
  torch::Tensor points = torch::zeros({nPoints, 3}, disps.options());
  torch::Tensor colors = torch::zeros({nPoints, 3}, images.options());
  torch::Tensor count = torch::zeros({nPoints}, disps.options());

  iproj_kernel<<<NUM_BLOCKS(ht * wd), THREADS>>>(index, nFrames, thresh,
    images.packed_accessor32<unsigned char,4,torch::RestrictPtrTraits>(),
    poses.packed_accessor32<float,2,torch::RestrictPtrTraits>(),
    disps.packed_accessor32<float,3,torch::RestrictPtrTraits>(),
    intrinsics.packed_accessor32<float,2,torch::RestrictPtrTraits>(),
    points.packed_accessor32<float,2,torch::RestrictPtrTraits>(),
    colors.packed_accessor32<unsigned char,2,torch::RestrictPtrTraits>(),
    count.packed_accessor32<float,1,torch::RestrictPtrTraits>());

  torch::Tensor m = masks[index].reshape({-1});
  torch::Tensor pointsFiltered, colorsFiltered;

  // std::cout << index << " " << dynamic << std::endl;

  pointsFiltered = torch::zeros({0, 3}, points.options());
  colorsFiltered = torch::zeros({0, 3}, colors.options());

  if (showForeground) {
    pointsFiltered = torch::cat({pointsFiltered, at::index(points, {(count >= 0) & (m < 0.5)})}, 0);
    colorsFiltered = torch::cat({colorsFiltered, at::index(colors, {(count >= 0) & (m < 0.5)})}, 0);
  }

  if (showBackground) {
    pointsFiltered = torch::cat({pointsFiltered, at::index(points, {(count >= 2.0) & (m > 0.5)})}, 0);
    colorsFiltered = torch::cat({colorsFiltered, at::index(colors, {(count >= 2.0) & (m > 0.5)})}, 0);
  }


  const int mPoints = pointsFiltered.size(0);

  return {mPoints, pointsFiltered, colorsFiltered};
}


__global__ void pose_to_matrix_kernel(
    const torch::PackedTensorAccessor32<float,2,torch::RestrictPtrTraits> poses,
    torch::PackedTensorAccessor32<float,3,torch::RestrictPtrTraits> mat4x4)
{

  const int index = blockIdx.x * THREADS + threadIdx.x;

  float t0[3], t[3];
  float q0[4], q[4];

  if (index < poses.size(0)) {

    t0[0] = poses[index][0];
    t0[1] = poses[index][1];
    t0[2] = poses[index][2];

    q0[0] = poses[index][3];
    q0[1] = poses[index][4];
    q0[2] = poses[index][5];
    q0[3] = poses[index][6];

    invSE3(t0, q0, t, q);

    mat4x4[index][0][0] = 1 - 2*q[1]*q[1] - 2*q[2]*q[2];
    mat4x4[index][0][1] = 2*q[0]*q[1] - 2*q[3]*q[2];
    mat4x4[index][0][2] = 2*q[0]*q[2] + 2*q[3]*q[1];
    mat4x4[index][0][3] = t[0];

    mat4x4[index][1][0] = 2*q[0]*q[1] + 2*q[3]*q[2];
    mat4x4[index][1][1] = 1 - 2*q[0]*q[0] - 2*q[2]*q[2];
    mat4x4[index][1][2] = 2*q[1]*q[2] - 2*q[3]*q[0];
    mat4x4[index][1][3] = t[1];

    mat4x4[index][2][0] = 2*q[0]*q[2] - 2*q[3]*q[1]; 
    mat4x4[index][2][1] = 2*q[1]*q[2] + 2*q[3]*q[0]; 
    mat4x4[index][2][2] = 1 - 2*q[0]*q[0] - 2*q[1]*q[1];
    mat4x4[index][2][3] = t[2];

    mat4x4[index][3][0] = 0.0;
    mat4x4[index][3][1] = 0.0;
    mat4x4[index][3][2] = 0.0;
    mat4x4[index][3][3] = 1.0;

  }
}


torch::Tensor poseToMatrix(const torch::Tensor poses) {
  const int num = poses.size(0);
  torch::Tensor mat4x4 = torch::zeros({num, 4, 4}, poses.options());

  pose_to_matrix_kernel<<<NUM_BLOCKS(num), THREADS>>>(
    poses.packed_accessor32<float,2,torch::RestrictPtrTraits>(),
    mat4x4.packed_accessor32<float,3,torch::RestrictPtrTraits>());

  return mat4x4;
}